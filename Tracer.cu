#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include <>

#include <iostream>
#include <fstream>

#include "vec3.h"
#include "color.h"
#include "ray.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__)

const char* ppm_filename = "Render.ppm";

/* Check if CUDA API call generated an error. Reset and exit if true. */
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << "at" <<
			file << ":" << line << " '" << func << "' \n";

		hipDeviceReset();
		exit(99);
	}
}

__device__ float hit_sphere(const point3& center, float radius, const ray& r)
{
	vec3 oc = r.origin() - center;
	float a = dot(r.direction(), r.direction());
	float b = 2.0f * dot(oc, r.direction());
	float c = dot(oc, oc) - radius * radius;
	float discriminant = b * b - 4.0f * a * c;
	if (discriminant < 0.0f) {
		return -1.0f;
	}
	else {
		return (-b - sqrt(discriminant)) / (2.0f * a);
	}
}

/* Based on the value of the y component in the normalized direction vector of the ray, calculate 
the final color by interpolating between white and color(0.5f, 0.7f, 1.0f). */
__device__ color ray_color(const ray& r)
{
	float t = hit_sphere(point3(0.0f, 0.0f, -1.0f), 0.5f, r);
	if (t > 0.0f) {
		vec3 N = unit_length(r.at(t) - vec3(0.0f, 0.0f, -1.0f));
		return 0.5f * color(N.x() + 1, N.y() + 1, N.z() + 1);
	}
	const vec3 unit_direction = unit_length(r.direction());
	t = 0.5f * (unit_direction.y() + 1.0f);
	return (1.0f - t) * color(1.0f, 1.0f, 1.0f) + t * color(0.5f, 0.7f, 1.0f);
}

__global__ void render(vec3 *fb, int max_x, int max_y, vec3 lower_left_corner, vec3 vertical, vec3 horizontal, vec3 origin)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	/* Early return if the current thread is not mapped to any pixel in the final render. */
	if ((i >= max_x) || (j >= max_y)) {
		return;
	}

	int pixel_index = j * max_x + i;
	float u = float(i) / float(max_x);
	float v = float(j) / float(max_y);
	ray r(origin, lower_left_corner + u * horizontal + v * vertical);
	fb[pixel_index] = ray_color(r);
}

int main(void)
{
	/* Image size. */
	const int nx = 1200, ny = 600;
	const int num_pixels = nx * ny;

	/* Camera properties. */
	const float viewport_height = 2.0f;
	const float viewport_width = (nx / ny) * viewport_height;
	const float focal_length = 1.0f;

	/* Viewport properties. */
	const vec3 origin = point3(0.0f, 0.0f, 0.0f);
	const vec3 horizontal = vec3(viewport_width, 0.0f, 0.0f);
	const vec3 vertical = vec3(0.0f, viewport_height, 0.0f);
	const vec3 lower_left_corner = origin - (horizontal / 2.0f) - (vertical / 2.0f) - vec3(0.0f, 0.0f, focal_length);

	/* Size of frame buffer in Unified memory to hold final pixel values. */
	size_t fb_size = num_pixels * sizeof(vec3);

	/* Allocate Unified memory for framebuffer */
	vec3* fb;
	checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

	/* Thread size for dividing work on GPU. */
	int tx = 8, ty = 8;

	/* Number of required blocks. */
	dim3 blocks(nx/tx+1, ny/ty+1);
	/* Number of threads per block. */
	dim3 threads(tx, ty);

	render<<<blocks, threads>>> (fb, nx, ny, lower_left_corner, vertical, horizontal, origin); 
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	/* Use fstream to write final pixel RGB values to the output ppm file. */
	std::ofstream out_ppm;
	out_ppm.open(ppm_filename);

	std::cout << "Writing to output file\n";
	/* Write the final pixel values from the buffer in Unified memory to the ppm output file. */
	out_ppm << "P3\n" << nx << " " << ny << "\n255\n";
	for (int j = ny - 1; j >= 0; j--) {
		for (int i = 0; i < nx; i++) {
			size_t pixel_index = j * nx + i;
			color pixel_color(fb[pixel_index].x(), fb[pixel_index].y(), fb[pixel_index].z());
			write_color(out_ppm, pixel_color);
		}
	}

	std::cout << "Done writing to output file\n";
	checkCudaErrors(hipFree(fb));

	out_ppm.close();

	return 0;
}